/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_bwd_kernel_zoh_s.cuh"
#include "selective_scan_bwd_kernel_zoh_f.cuh"
#include "selective_scan_bwd_kernel_foh_s.cuh"
#include "selective_scan_bwd_kernel_foh_f.cuh"

template void selective_scan_bwd_zohs_cuda<float, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_zohf_cuda<float, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_fohs_cuda<float, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_fohf_cuda<float, float>(SSMParamsBwd &params, hipStream_t stream);