/******************************************************************************
 * Copyright (c) 2024, Yujie Zhu.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_fwd_kernel_zoh_s.cuh"
#include "selective_scan_fwd_kernel_zoh_f.cuh"
#include "selective_scan_fwd_kernel_foh_s.cuh"
#include "selective_scan_fwd_kernel_foh_f.cuh"

template void selective_scan_fwd_zohs_cuda<at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_zohs_cuda<at::BFloat16, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_zohf_cuda<at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_zohf_cuda<at::BFloat16, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohs_cuda<at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohs_cuda<at::BFloat16, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohf_cuda<at::BFloat16, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohf_cuda<at::BFloat16, complex_t>(SSMParamsBase &params, hipStream_t stream);