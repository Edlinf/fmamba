/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_bwd_kernel_zoh_s.cuh"
#include "selective_scan_bwd_kernel_zoh_f.cuh"
#include "selective_scan_bwd_kernel_foh_s.cuh"
#include "selective_scan_bwd_kernel_foh_f.cuh"

template void selective_scan_bwd_zohs_cuda<at::BFloat16, complex_t>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_zohf_cuda<at::BFloat16, complex_t>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_fohs_cuda<at::BFloat16, complex_t>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_fohf_cuda<at::BFloat16, complex_t>(SSMParamsBwd &params, hipStream_t stream);