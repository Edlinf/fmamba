/******************************************************************************
 * Copyright (c) 2024, Yujie Zhu.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_fwd_kernel_zoh_s.cuh"
#include "selective_scan_fwd_kernel_zoh_f.cuh"
#include "selective_scan_fwd_kernel_foh_s.cuh"
#include "selective_scan_fwd_kernel_foh_f.cuh"

template void selective_scan_fwd_zohs_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_zohs_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_zohf_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_zohf_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohs_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohs_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohf_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_fohf_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);